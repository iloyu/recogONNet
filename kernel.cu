#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"iostream"
#include <stdio.h>
#include "../common/book.h"
#include"time.h"
#define SIZE    (100*1024*1024)

int main( void ) {
    unsigned char *buffer =
                     (unsigned char*)big_random_block( SIZE );

    // capture the start time
    clock_t         start, stop;
    start = clock();

    unsigned int    histo[256];
    for (int i=0; i<256; i++)
        histo[i] = 0;

    for (int i=0; i<SIZE; i++)
        histo[buffer[i]]++;

    stop = clock();
    float   elapsedTime = (float)(stop - start) /
                          (float)CLOCKS_PER_SEC * 1000.0f;
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    long histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum:  %ld\n", histoCount );

    free( buffer );
	getchar();
    return 0;
}
