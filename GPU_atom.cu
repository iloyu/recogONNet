#include "hip/hip_runtime.h"

#include <stdio.h>
#include "../common/book.h"

#include "hip/hip_runtime.h"
#include ""
#define SIZE    (100*1024*1024)


__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {
    // calculate the starting index and the offset to the next
    // block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        
        atomicAdd( &histo[buffer[i]], 1 );
        i += stride;
    }
}

int main( void ) {
    unsigned char *buffer =
                     (unsigned char*)big_random_block( SIZE );

    // capture the start time
    // starting the timer here so that we include the cost of
    // all of the operations on the GPU.
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // allocate memory on the GPU for the file's data
    unsigned char *dev_buffer;
    unsigned int *dev_histo;
    HANDLE_ERROR( hipMalloc( (void**)&dev_buffer, SIZE ) );
    HANDLE_ERROR( hipMemcpy( dev_buffer, buffer, SIZE,
                              hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMalloc( (void**)&dev_histo,
                              256 * sizeof( int ) ) );
    HANDLE_ERROR( hipMemset( dev_histo, 0,
                              256 * sizeof( int ) ) );

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t  prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2,256>>>( dev_buffer, SIZE, dev_histo );
    
    unsigned int    histo[256];
    HANDLE_ERROR( hipMemcpy( histo, dev_histo,
                              256 * sizeof( int ),
                              hipMemcpyDeviceToHost ) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    long histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum:  %ld\n", histoCount );

    // verify that we have the same counts via CPU
    for (int i=0; i<SIZE; i++)
        histo[buffer[i]]--;
    for (int i=0; i<256; i++) {
        if (histo[i] != 0)
            printf( "Failure at %d!  Off by %d\n", i, histo[i] );
    }

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    hipFree( dev_histo );
    hipFree( dev_buffer );
    free( buffer );
    getchar();
    return 0;
}
